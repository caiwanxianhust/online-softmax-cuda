#include "hip/hip_runtime.h"
#include "online_softmax.cuh"
#include "utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

namespace softmax
{
    template <typename T>
    struct MaxOp
    {
        __device__ __forceinline__ T operator()(const T &a, const T &b) { return max(a, b); }
    };

    template <typename T>
    struct SumOp
    {
        __device__ __forceinline__ T operator()(const T &a, const T &b) { return a + b; }
    };

    template <template <typename> class ReduceOp, typename T>
    __device__ __inline__ T warpAllReduce(T val)
    {
        auto functor = ReduceOp<T>();
#pragma unroll
        for (int mask = 16; mask > 0; mask >>= 1)
        {
            val = functor(val, __shfl_xor_sync(0xffffffff, val, mask, 32));
        }
        return val;
    }

    template <typename T>
    __device__ __inline__ T blockAllReduceSum(T val)
    {
        __shared__ T shared[32];
        __shared__ T ret;
        int warp_id = (threadIdx.x >> 5);
        int lane_id = (threadIdx.x & 31);

        val = warpAllReduce<SumOp, T>(val);
        if (lane_id == 0) { shared[warp_id] = val; }
        __syncthreads();

        val = (threadIdx.x < (blockDim.x >> 5)) ? shared[threadIdx.x] : (T)(0.0f);
        val = warpAllReduce<SumOp, T>(val);
        if (threadIdx.x == 0) { ret = val; }
        __syncthreads();

        return ret;
    }

    template <typename T>
    __device__ __inline__ T blockAllReduceMax(T val)
    {
        __shared__ T shared[32];
        __shared__ T ret;
        int warp_id = (threadIdx.x >> 5);
        int lane_id = (threadIdx.x & 31);

        val = warpAllReduce<MaxOp, T>(val);
        if (lane_id == 0) { shared[warp_id] = val; }
        __syncthreads();

        val = (threadIdx.x < (blockDim.x >> 5)) ? shared[threadIdx.x] : (T)(- FLT_MAX);
        val = warpAllReduce<MaxOp, T>(val);
        if (threadIdx.x == 0) { ret = val; }
        __syncthreads();

        return ret;
    }


    __global__ void softmaxKernel(const float *__restrict__ mat, float *__restrict__ output, const int ncol)
    {
        float val;
        float vmax = - FLT_MAX;
        float exp_sum = 1e-10f;

        #pragma unroll
        for (int i = threadIdx.x; i < ncol; i += blockDim.x)
        {
            vmax = max(mat[blockIdx.x * ncol + i], vmax);
        }
        __syncthreads();

        vmax = blockAllReduceMax<float>(vmax);

        #pragma unroll
        for (int i = threadIdx.x; i < ncol; i += blockDim.x)
        {
            exp_sum += __expf(mat[blockIdx.x * ncol + i] - vmax); 
        }
        __syncthreads();

        exp_sum = blockAllReduceSum<float>(exp_sum);

        #pragma unroll
        for (int i = threadIdx.x; i < ncol; i += blockDim.x)
        {
            val = __expf(mat[blockIdx.x * ncol + i] - vmax) / exp_sum; 
            output[blockIdx.x * ncol + i] = val;
        }
    }

    void launchSoftmaxKernel(const float *__restrict__ mat, float *__restrict__ output, const int ncol, const int nrow, hipStream_t stream)
    {
        dim3 block(256);
        dim3 grid(nrow);
        softmaxKernel<<<grid, block, 0, stream>>>(mat, output, ncol);
    }

} // namespace softmax